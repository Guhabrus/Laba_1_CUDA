#include "hip/hip_runtime.h"
﻿
#include <stdio.h>
#include <stdlib.h>
#include <float.h>
#include <time.h>

#include "hip/hip_runtime.h"
#include ""

#define MASKA_W  3
#define RADIUS MASKA_W/2
#define TILE_WIDTH 16
#define w (TILE_WIDTH + MASKA_W - 1)
#define clamp(x) (min(max((x), 0.0), 1.0))



#define MAX_IMAGE_WIDTH  3840
#define MAX_IMAGEHEIGHT  2160
#define MAX_GRAY_LEVEL  255      
#define GRAYLEVEL       256      
#define MAX_FILENAME    256      
#define MAX_BUFFERSIZE  256


float image1[MAX_IMAGE_WIDTH][MAX_IMAGEHEIGHT],
image2[MAX_IMAGE_WIDTH][MAX_IMAGEHEIGHT];
int x_size1, y_size1, x_size2, y_size2;




void load_image_data()
{
    char file_name[MAX_FILENAME];
    char buffer[MAX_BUFFERSIZE];
    FILE* fp;
    int max_gray;
    int x, y;


    //printf("<Файл должен быть в формате .pgm>\n\n");
    printf("Name? (*.pgm) : ");
    scanf("%s", file_name);
    fp = fopen(file_name, "rb");
    if (NULL == fp) {
        printf(" not such file in directory!\n\n");
        exit(1);
    }
    /* Check of file-type ---P5 */
    fgets(buffer, MAX_BUFFERSIZE, fp);
    if (buffer[0] != 'P' || buffer[1] != '5') {
        printf(" Wrong format (нужен .pgm)!\n\n");
        exit(1);
    }
    /* input of x_size1, y_size1 */
    x_size1 = 0;
    y_size1 = 0;
    while (x_size1 == 0 || y_size1 == 0) {
        fgets(buffer, MAX_BUFFERSIZE, fp);
        if (buffer[0] != '#') {
            sscanf(buffer, "%d %d", &x_size1, &y_size1);
        }
    }
    /* input of max_gray */
    max_gray = 0;
    while (max_gray == 0) {
        fgets(buffer, MAX_BUFFERSIZE, fp);
        if (buffer[0] != '#') {
            sscanf(buffer, "%d", &max_gray);
        }
    }
    /* Display of parameters */

    if (x_size1 > MAX_IMAGE_WIDTH || y_size1 > MAX_IMAGEHEIGHT) {
        printf("     wrong size, need - %d x %d\n\n", MAX_IMAGE_WIDTH, MAX_IMAGEHEIGHT);
        exit(1);
    }
    if (max_gray != MAX_GRAY_LEVEL) {
        printf("    yarko!\n\n");
        exit(1);
    }
    /* Input of image data*/
    for (y = 0; y < y_size1; y++) {
        for (x = 0; x < x_size1; x++) {
            image1[y][x] = (unsigned char)fgetc(fp);
        }
    }
    printf("----Succes-----\n\n");
    fclose(fp);
}




void save_image_data()
{
    char file_name[MAX_FILENAME];
    FILE* fp; /* File pointer */
    int x, y; /* Loop variable */

    /* Output file open */


    printf("<format file .pgm>\n\n");
    printf("Name output file? (*.pgm) : ");
    scanf("%s", file_name);
    fp = fopen(file_name, "wb");
    /* output of pgm file header information */
    fputs("P5\n", fp);
    fputs("# Created by Image Processing\n", fp);
    fprintf(fp, "%d %d\n", x_size2, y_size2);
    fprintf(fp, "%d\n", MAX_GRAY_LEVEL);
    /* Output of image data */
    for (y = 0; y < y_size2; y++) {
        for (x = 0; x < x_size2; x++) {
            fputc(image2[y][x], fp);
        }
    }
    printf("\n-----Image data output OK-----\n\n");

    fclose(fp);
}



void load_image_file(char* filename)
{
    char buffer[MAX_BUFFERSIZE];
    FILE* fp;
    int max_gray;
    int x, y;

    /* Input file open */
    fp = fopen(filename, "rb");
    if (NULL == fp) {
        printf("     The file doesn't exist!\n\n");
        exit(1);
    }
    /* Check of file-type ---P5 */
    fgets(buffer, MAX_BUFFERSIZE, fp);
    if (buffer[0] != 'P' || buffer[1] != '5') {
        printf("     Mistaken file format, not P5!\n\n");
        exit(1);
    }
    /* input of x_size1, y_size1 */
    x_size1 = 0;
    y_size1 = 0;
    while (x_size1 == 0 || y_size1 == 0) {
        fgets(buffer, MAX_BUFFERSIZE, fp);
        if (buffer[0] != '#') {
            sscanf(buffer, "%d %d", &x_size1, &y_size1);
        }
    }
    /* input of max_gray */
    max_gray = 0;
    while (max_gray == 0) {
        fgets(buffer, MAX_BUFFERSIZE, fp);
        if (buffer[0] != '#') {
            sscanf(buffer, "%d", &max_gray);
        }
    }
    if (x_size1 > MAX_IMAGE_WIDTH || y_size1 > MAX_IMAGEHEIGHT) {
        printf("     Image size exceeds %d x %d\n\n",
            MAX_IMAGE_WIDTH, MAX_IMAGEHEIGHT);
        printf("     Please use smaller images!\n\n");
        exit(1);
    }
    if (max_gray != MAX_GRAY_LEVEL) {
        printf("     Invalid value of maximum gray level!\n\n");
        exit(1);
    }
    /* Input of image data*/
    for (y = 0; y < y_size1; y++) {
        for (x = 0; x < x_size1; x++) {
            image1[y][x] = (float)fgetc(fp);
        }
    }
    fclose(fp);
}




void save_image_file(char* filename)
{
    FILE* point_file;
    int x, y;

    point_file = fopen(filename, "wb");

    fputs("P\n", point_file);
    fputs("Processing\n", point_file);
    fprintf(point_file, "%d %d\n", x_size2, y_size2);
    fprintf(point_file, "%d\n", MAX_GRAY_LEVEL);
    /* Output of image data */
    for (y = 0; y < y_size2; y++) {
        for (x = 0; x < x_size2; x++) {
            fputc(image2[y][x], point_file);
        }
    }
    fclose(point_file);
}




__global__ void svertka(float* I, const float* __restrict__ M, float* P, int width, int height) {
    __shared__ float N_ds[w][w];


    // Загрузка первой партии
    int dest = threadIdx.y * TILE_WIDTH + threadIdx.x, destY = dest / w, destX = dest % w,
        srcY = blockIdx.y * TILE_WIDTH + destY - RADIUS, srcX = blockIdx.x * TILE_WIDTH + destX - RADIUS,
        src = srcY * width + srcX;

    if (srcY >= 0 && srcY < height && srcX >= 0 && srcX < width)
    {
        N_ds[destY][destX] = I[src];
    }
    else
    {
        N_ds[destY][destX] = 0;
    }

    for (int i = 1; i <= (w * w) / (TILE_WIDTH * TILE_WIDTH); i++)
    {
        // Загрузка вторйо партии
        dest = threadIdx.y * TILE_WIDTH + threadIdx.x + TILE_WIDTH * TILE_WIDTH;
        destY = dest / w, destX = dest % w;
        srcY = blockIdx.y * TILE_WIDTH + destY - RADIUS;
        srcX = blockIdx.x * TILE_WIDTH + destX - RADIUS;
        src = srcY * width + srcX;
        if (destY < w) {
            if (srcY >= 0 && srcY < height && srcX >= 0 && srcX < width)
                N_ds[destY][destX] = I[src];
            else
                N_ds[destY][destX] = 0;
        }
    }
    __syncthreads();

    float iac_cum = 0;
    int y, x;
    for (y = 0; y < MASKA_W; y++)
    {
        for (x = 0; x < MASKA_W; x++)
        {
            iac_cum += N_ds[threadIdx.y + y][threadIdx.x + x] * M[y * MASKA_W + x];
        }

    }


    y = blockIdx.y * TILE_WIDTH + threadIdx.y;
    x = blockIdx.x * TILE_WIDTH + threadIdx.x;
    if (y < height && x < width)
        P[y * width + x] = iac_cum;

    __syncthreads();

}

void sobel_filtering()
{

    float weight[3][3] = { { -1,  0,  1 },
                           { -2,  0,  2 },
                           { -1,  0,  1 } };
    float pixel_value;

    int x, y, i, j;
    float* Image_input_from_device_point;
    float* Image_output_from_device_poin;
    float* deviceMask;

    hipMalloc((void**)&Image_input_from_device_point, x_size1 * y_size1 * sizeof(float));
    hipMalloc((void**)&Image_output_from_device_poin, x_size1 * y_size1 * sizeof(float));
    hipMalloc((void**)&deviceMask, 3 * 3 * sizeof(float));

    hipMemcpy(Image_input_from_device_point, image1, x_size1 * y_size1 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(deviceMask, weight, 3 * 3 * sizeof(float), hipMemcpyHostToDevice);



    x_size2 = x_size1;
    y_size2 = y_size1;
    for (y = 0; y < y_size2; y++)
    {
        for (x = 0; x < x_size2; x++)
        {
            image2[y][x] = 0;
        }
    }

    dim3 dimGrid(ceil((float)x_size1 / TILE_WIDTH), ceil((float)y_size1 / TILE_WIDTH));
    dim3 dimBlock(TILE_WIDTH, TILE_WIDTH);
    svertka << <dimGrid, dimBlock >> > (Image_input_from_device_point, deviceMask, Image_output_from_device_poin, x_size1, y_size1);


    hipMemcpy(image2, Image_output_from_device_poin, x_size2 * y_size2 * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(Image_input_from_device_point);
    hipFree(Image_output_from_device_poin);
    hipFree(deviceMask);

}


int main()
{

    
    load_image_data();

    clock_t begin = clock();
    sobel_filtering();   //Приминение фильтра собеля
    clock_t end = clock();
    double time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
    printf("\n\nTime: %f\n", time_spent);
    save_image_data();
    return 0;
}